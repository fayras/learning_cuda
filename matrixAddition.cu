#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <math.h>

struct Matrix {
	int rows;
	int cols;
	float* data;
};

Matrix* createMatrix(int rows, int columns) {
	int size = rows * columns;

	Matrix* m = (Matrix*) malloc(sizeof(Matrix));
	m->rows = rows;
	m->cols = columns;
	m->data = (float*) malloc(size * sizeof(float));

	return m;
}

Matrix* createRandomMatrix(int rows, int columns) {
	Matrix* m = createMatrix(rows, columns);

	for (int i = 0; i < rows * columns; i++) {
		m->data[i] = ((float)rand() / (float)(RAND_MAX)) * 5;
	}

	return m;
}

void printMatrix(Matrix* m) {
	for (int i = 0; i < m->rows; i++) {
		for (int j = 0; j < m->cols; j++) {
			printf("%.2f ", m->data[i * m->cols + j]);
		}
		printf("\n");
	}
	printf("\n");
}

Matrix* addOnCPU(Matrix* m1, Matrix* m2) {
	if (m1->rows != m2->rows || m1->cols != m2->cols) {
		return NULL;
	}

	int size = m1->rows * m1->cols;
	Matrix* m3 = createMatrix(m1->rows, m1->cols);

	for (int i = 0; i < size; i++) {
		m3->data[i] = m1->data[i] + m2->data[i];
	}

	return m3;
}

__global__ void kernel(float* m1, float* m2, float* m3, const int width, const int total) {
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	int index = row * width + col;

	if (index < total) {
		m3[index] = m1[index] + m2[index];
	}
}

Matrix* addOnGPU(Matrix* m1, Matrix* m2, dim3 grid, dim3 block) {
	if (m1->rows != m2->rows || m1->cols != m2->cols) {
		return NULL;
	}

	int size = m1->rows * m1->cols * sizeof(float);
	float* d_m1;
	float* d_m2;
	float* d_m3;

	hipMalloc((void**)&d_m1, size);
	hipMemcpy(d_m1, m1->data, size, hipMemcpyHostToDevice);
	hipMalloc((void**)&d_m2, size);
	hipMemcpy(d_m2, m2->data, size, hipMemcpyHostToDevice);
	hipMalloc((void**)&d_m3, size);

	kernel <<<grid, block>>> (d_m1, d_m2, d_m3, m1->cols, m1->rows * m1->cols);

	Matrix* m3 = createMatrix(m1->rows, m2->cols);

	hipMemcpy(m3->data, d_m3, size, hipMemcpyDeviceToHost);
	hipFree(d_m1);
	hipFree(d_m2);
	hipFree(d_m3);

	return m3;
}

bool checkResult(Matrix* hostMatrix, Matrix* deviceMatrix) {
	double e = 1.0E-8;
	int size = hostMatrix->rows * hostMatrix->cols;

	if (hostMatrix->rows != deviceMatrix->rows || hostMatrix->cols != deviceMatrix->cols) {
		return false;
	}

	for (int i = 0; i < size; i++) {
		if (abs((long)(hostMatrix->data[i] - deviceMatrix->data[i])) > e) {
			printf("Matrices do not match!\n");
			printf("Index %d \n", i);
			printf("CPU: %.2f, GPU: %.2f \n", hostMatrix->data[i], deviceMatrix->data[i]);
			return false;
		}
	}

	return true;
}

#define BLOCK_WIDTH_X 16
#define BLOCK_WIDTH_Y 16

int main() {
	const int dimX = 1000;
	const int dimY = 1000;
	
	dim3 dimBlock(BLOCK_WIDTH_X, BLOCK_WIDTH_Y);
	dim3 dimGrid(ceil(dimX / (float)BLOCK_WIDTH_X), ceil(dimY / (float)BLOCK_WIDTH_Y));

	Matrix* m1 = createRandomMatrix(dimX, dimY);
	Matrix* m2 = createRandomMatrix(dimX, dimY);

	Matrix* m3 = addOnCPU(m1, m2);
	Matrix* m4 = addOnGPU(m1, m2, dimGrid, dimBlock);

	bool same = checkResult(m3, m4);

	// printMatrix(m1);
	// printMatrix(m2);
	// printMatrix(m3);
	// printMatrix(m4);

	free(m1->data);
	free(m2->data);
	free(m3->data);
	free(m4->data);
	free(m1);
	free(m2);
	free(m3);
	free(m4);

	hipDeviceReset();

	if (!same) {
		return 1;
	}

	return 0;
}